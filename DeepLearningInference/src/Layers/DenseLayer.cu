#include "DenseLayer.cuh"
#include "../cuBLAS/blasUtills.cuh"

DenseLayer::DenseLayer(int size, Activation activationFunc, std::string name) : Layer(std::vector<int>(1, size), name) {
	this->activationFunc = activationFunc;;
}

LayerType DenseLayer::getLayerType() {
	return LayerType::DENSE;
}


void* DenseLayer::allocAndInitDataOnDevice(void* inputDataOnHost, int inputElementCount, std::list<Layer*>::iterator layerIterator) {

	Layer* prevLayer = *(std::prev(layerIterator, 1));

	float* inputVectorOnHost = (float*)inputDataOnHost;
	int inputSize = prevLayer->getSize().front();
	float* inputVectorOnDevice; // n x 1
	hipMallocManaged((void**)&inputVectorOnDevice, this->getSize().front() * 1 * sizeof(float));
	for (size_t i = 0; i < inputSize; i++)
	{
		inputVectorOnDevice[i] = inputVectorOnHost[i];
	}
	this->outputOfCurrentLayer = inputVectorOnDevice;

	return this->outputOfCurrentLayer;
}


// Allocates host memory for holding the weights and bias pertaining to links from previous layer to the current layer
void DenseLayer::init() {
	this->weights = std::vector<std::vector<float>>(0);
	this->bias = std::vector<float>(0);
}

void* DenseLayer::getOuputOnDevice() {
	return (void*)(this->outputOfCurrentLayer);
}

void DenseLayer::initWeight(const std::vector<float> &weights) {
	int numberOfRows = this->getSize()[0];
	int numberOfCols = weights.size() / numberOfRows;
	int startIndexOfIthRow = 0;
	for (int i = 0; i < numberOfRows; i++) {
		std::vector<float> row(weights.begin() + startIndexOfIthRow, weights.begin() + startIndexOfIthRow + numberOfCols);
		this->weights.push_back(row);
		startIndexOfIthRow += numberOfCols;
	}
}

void DenseLayer::initBias(const std::vector<float>& bias) {
	this->bias = bias;
}

void DenseLayer::forward(ContextFactory contextFactory, std::vector<float>& input_sample) {
	//Z = W * X + B
	BlasUtils::axpby_vector_matrix(contextFactory, this->weights, input_sample, this->bias);
	//	A = f(Z)
	BlasUtils::computeActivation(input_sample, this->activationFunc);
}

void DenseLayer::forward(ContextFactory contextFactory, Layer* previousLayer) {
	
	float* outputFromPreviousLayer = (float*)previousLayer->getOuputOnDevice();
	int outputSizeFromPreviousLayer = previousLayer->getSize().front();
	
	float* outputOfCurrentLayer;
	int outputSizeOfCurrentLayer = this->getSize().front();
	
	//Z = W * X + B
	BlasUtils::axpby_vector_matrix(contextFactory,
		outputFromPreviousLayer, outputSizeFromPreviousLayer,
		this->weights, this->bias, outputOfCurrentLayer, outputSizeOfCurrentLayer);

	//	A = f(Z)
	BlasUtils::computeActivation(outputOfCurrentLayer, outputSizeOfCurrentLayer, this->activationFunc);

	this->outputOfCurrentLayer = outputOfCurrentLayer;
	for (int i = 0; i < outputSizeOfCurrentLayer; i++)
		std::cout << outputOfCurrentLayer[i] << " ";
	std::cout << "\n";
}

void DenseLayer::forward(ContextFactory contextFactory, void* inputSample, int inputElementCount, std::list<Layer*>::iterator layerIterator) {
	
	this->allocAndInitDataOnDevice(inputSample, inputElementCount, layerIterator);
	
	Layer* prevLayer = *(std::prev(layerIterator, 1));
	float* outputFromPreviousLayer = (float*)prevLayer->getOuputOnDevice();
	int outputSizeFromPreviousLayer = prevLayer->getSize().front();
	
	float* outputForCurrentLayer;
	int outputSizeForCurrentLayer;

	//Z = W * X + B
	BlasUtils::axpby_vector_matrix(contextFactory,
		outputFromPreviousLayer, outputSizeFromPreviousLayer,
		this->weights, this->bias,
		outputForCurrentLayer, outputSizeForCurrentLayer
	);

	//	A = f(Z)
	BlasUtils::computeActivation(outputForCurrentLayer, outputSizeForCurrentLayer, this->activationFunc);

	this->outputOfCurrentLayer = outputOfCurrentLayer;
}