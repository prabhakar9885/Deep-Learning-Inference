#include "./ContextObject.cuh"

ContextObject::ContextObject()
{
	cublas_handle = nullptr;
	cudnn_handle = nullptr;
}

hipblasHandle_t* ContextObject::getCublasHandle()
{
	if (this->cublas_handle == nullptr)
	{
		this->cublas_handle = new hipblasHandle_t();
		hipblasStatus_t status;
		if ((status = hipblasCreate(this->cublas_handle)) != HIPBLAS_STATUS_SUCCESS)
		{
			throw "cuBLAS initialization Failed. Status code: " + std::to_string(status) + ".";
		}
	}
	return this->cublas_handle;
}


hipdnnHandle_t* ContextObject::getCudnnHandle()
{
	if (this->cudnn_handle == nullptr)
	{
		this->cudnn_handle = new hipdnnHandle_t();
		hipdnnStatus_t status;
		if ((status = hipdnnCreate(this->cudnn_handle)) != HIPDNN_STATUS_SUCCESS)
		{
			throw "cuDNN initialization Failed. Status code: " + std::to_string(status) + ".";
		}
	}
	return this->cudnn_handle;
}


void ContextObject::releaseCublasHandle()
{
	if (cublas_handle == nullptr)
		return;

	hipblasStatus_t status = hipblasDestroy(*cublas_handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		throw "Failed to release the cuBLAS handle. Status code: " + std::to_string(status) + ".";
	}
	free(this->cublas_handle);
}


void ContextObject::releaseCudnnHandle()
{
	if (cudnn_handle == nullptr)
		return;

	hipdnnStatus_t status = hipdnnDestroy(*cudnn_handle);
	if (status != HIPDNN_STATUS_SUCCESS)
	{
		throw "Failed to release the cuDNN handle. Status code: " + std::to_string(status) + ".";
	}
	free(this->cudnn_handle);
}