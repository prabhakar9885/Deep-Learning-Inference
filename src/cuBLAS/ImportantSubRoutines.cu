#include "hip/hip_runtime.h"
// This program shows off some basic cuBLAS examples
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <algorithm> 
#include "iostream"
#include "iomanip"

using namespace std;

#pragma region Error handling code

inline hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

#pragma endregion


#pragma region Auxilary functions for Matrix and vectors

int IDX2C(int i, int j, int ld)
{
	return j * ld + i;
}

void vector_init(float* a, int startValue, int n)
{
	for (int i = 0; i < n; i++)
	{
		a[i] = startValue++;
	}
}

void matrix_init(float* mat, int startValue, int m, int n, int ld)
{
	int val = startValue;
	for (int j = 0; j < n; j++)	// Columns
	{
		for (int i = 0; i < m; i++)  // Rows
		{
			mat[IDX2C(i, j, ld)] = val;
			val++;
		}
	}
}

void matrix_init_all(float* mat, int value, int m, int n, int ld)
{
	for (int j = 0; j < n; j++)	// Columns
	{
		for (int i = 0; i < m; i++)  // Rows
		{
			mat[IDX2C(i, j, ld)] = value;
		}
	}
}

void display_matrix(float* mat, int m, int n, int ld)
{
	for (int i = 0; i < m; i++) // Rows
	{
		for (int j = 0; j < n; j++) // Columns
		{
			cout << mat[IDX2C(i, j, ld)] << std::setw(5);
		}
		cout << "\n";
	}
	cout << "\n";
}

void display_vector(float* vec, int n) {
	for (int i = 0; i < n; i++)
		cout << vec[i] << std::setw(3);
}

#pragma endregion


#pragma region Kernals

__global__ void sigmoid(float* inp, float* res, int N) {
	int indx = blockDim.x * blockIdx.x + threadIdx.x;
	if (indx < N)
		res[indx] = 1 / (1 + exp(-inp[indx]));
}


#pragma endregion


#pragma region Linear algebra functions

void mat_mul() {
	cout << "Mat mul" << "\n" << "===============" << "\n";

	int p = 6, q = 4, r = 2;
	float* mat1; //  Human rep: p x q
	float* mat2; //  Human rep: q x r
	float* mat_prod;
	float* tmp;
	hipError_t err;

	err = hipMallocManaged((void**)&mat1, sizeof(float) * p * q);
	checkCuda(err);
	err = hipMallocManaged((void**)&mat2, sizeof(float) * q * r);
	checkCuda(err);
	err = hipMallocManaged((void**)&mat_prod, sizeof(float) * p * r);
	checkCuda(err);

	matrix_init(mat1, 0, p, q, p);
	matrix_init(mat2, 0, q, r, q);

	// Scalaing factors
	float alpha = 1.0f;
	float beta = 0.0f;

	// Create and initialize a new context
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);

	status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, p, r, q, &alpha, mat1, p, mat2, q, &beta, mat_prod, p);

	hipDeviceSynchronize();

	display_matrix(mat1, p, q, p);
	display_matrix(mat2, q, r, q);
	display_matrix(mat_prod, r, p, r);

	// Clean up the created handle
	hipblasDestroy(handle);

	// Release allocated memory
	hipFree(mat1);
	hipFree(mat2);
	hipFree(mat_prod);

	cout << "\n\n";
}

void axpy() {

	cout << "a*x + y" << "\n";
	cout << "=========================" << "\n\n";

	int p = 6;
	float* x;
	float* y;
	float alpha = 2.0f;
	hipError_t err;

	err = hipMallocManaged((void**)&x, sizeof(float) * p);
	checkCuda(err);
	err = hipMallocManaged((void**)&y, sizeof(float) * p);
	checkCuda(err);

	vector_init(x, 0, p);
	vector_init(y, 2, p);
	cout << "x : ";	display_vector(x, p); cout << "\n";
	cout << "y : ";	display_vector(y, p); cout << "\n";

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t status = hipblasSaxpy(handle, p, &alpha, x, 1, y, 1);
	hipDeviceSynchronize();

	cout << "y : ";	display_vector(y, p); cout << "\n";

	hipblasDestroy(handle);
	hipFree(x);
	hipFree(y);

	cout << "=========================" << "\n\n";
}

void map() {
	float* arr;
	float* res;
	int N = 10;

	hipMallocManaged(&arr, N * sizeof(float));
	hipMallocManaged(&res, N * sizeof(float));

	for (int i = 0; i < N; i++)
		arr[i] = N / 2 - i;

	int blockSize = 8;
	int blockCount = N / blockSize + (N % blockSize != 0);

	sigmoid << < blockCount, blockSize >> > (arr, res, N);
	hipDeviceSynchronize();

	cout << "First 5 elements\n";
	for (int i = 0; i < min(5, N); i++)
		cout << "sigmoid(" << arr[i] << ") : " << res[i] << "\n";

	cout << "Last 5 elements\n";
	for (int i = max(N - 5, 0); i < max(5, N); i++)
		cout << "sigmoid(" << arr[i] << ") : " << res[i] << "\n";

	hipFree(arr);
	hipFree(res);
}

void axpb_vector_matrix() {

	size_t m = 5, n = 3;
	float* A; // m x n
	float* X; // n x 1
	float b = 1; // scalar
	float* C; // m x 1
	float alpha = 1;

	hipMallocManaged(&A, m * n * sizeof(float));
	hipMallocManaged(&X, n * 1 * sizeof(float));
	hipMallocManaged(&C, m * 1 * sizeof(float));

	matrix_init(A, 0, m, n, m);
	matrix_init(X, 0, n, 1, n);
	matrix_init_all(C, 1, m, 1, m);

	display_matrix(A, m, n, m);
	display_matrix(X, n, 1, n);
	display_matrix(C, m, 1, m);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, 1, n, &alpha, A, m, X, n, &b, C, m);
	hipDeviceSynchronize();

	display_matrix(C, m, 1, m);

	hipblasDestroy(handle);
	hipFree(A);
	hipFree(X);
	hipFree(C);
}

void transpose_mat() {

	cout << "Transpose of a matrix\n";
	cout << "=========================" << "\n";
	float alpha = 1;
	float beta = 0;
	float* mat;
	float* res;
	size_t r = 3, c = 2;

	hipMallocManaged(&mat, r * c * sizeof(float));
	hipMallocManaged(&res, c * r * sizeof(float));

	matrix_init(mat, 0, r, c, r);
	display_matrix(mat, r, c, r);

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, c, r, &alpha, mat, r, &beta, mat, r, res, c);
	hipDeviceSynchronize();
	display_matrix(res, c, r, c);

	hipblasDestroy(handle);
	hipFree(mat);
	hipFree(res);
}

#pragma endregion


/**
 *	https://www.adityaagrawal.net/blog/deep_learning/row_column_major
 */
void mat_mul_2() {
	cout << "Mat mul" << "\n" << "===============" << "\n";

	int p = 6, q = 4, r = 2;
	float* mat1; //  Human rep: p x q
	float* mat2; //  Human rep: q x r
	float* mat_prod;
	float* mat_prod_T;
	float* tmp;
	hipError_t err;

	err = hipMallocManaged((void**)&mat1, sizeof(float) * p * q);
	checkCuda(err);
	err = hipMallocManaged((void**)&mat2, sizeof(float) * q * r);
	checkCuda(err);
	err = hipMallocManaged((void**)&mat_prod, sizeof(float) * p * r);
	checkCuda(err);
	err = hipMallocManaged((void**)&mat_prod_T, sizeof(float) * p * r);
	checkCuda(err);

	// Init mat1
	int val = 0;
	for (size_t i = 0; i < p; i++)
	{
		for (size_t j = 0; j < q; j++)
		{
			mat1[i * q + j] = val;
			val = (val + 1) % 5;
		}
	}

	for (size_t i = 0; i < p; i++)
	{
		for (size_t j = 0; j < q; j++)
		{
			cout << mat1[i * q + j] << std::setw(5);
		}
		cout << "\n";
	}
	cout << "\n";

	// Init mat2
	val = 0;
	for (size_t i = 0; i < q; i++)
	{
		for (size_t j = 0; j < r; j++)
		{
			mat2[i * r + j] = val;
			val = (val + 1) % 5;
		}
	}
	for (size_t i = 0; i < q; i++)
	{
		for (size_t j = 0; j < r; j++)
		{
			cout << mat2[i * r + j] << std::setw(5);
		}
		cout << "\n";
	}
	cout << "\n";

	// Scalaing factors
	float alpha = 1.0f;
	float beta = 0.0f;

	// Create and initialize a new context
	hipblasHandle_t handle;
	hipblasStatus_t status;
	hipblasCreate(&handle);

	//	CMO => mat_prod := alpha * mat2.T * mat1.T + beta * mat_prod.T
	//					:= mat2.T * mat1.T
	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, r, p, q, &alpha, mat2, r, mat1, q, &beta, mat_prod, r);
	hipDeviceSynchronize();
	for (size_t i = 0; i < r; i++)
	{
		for (size_t j = 0; j < p; j++)
		{
			cout << mat_prod[i * p + j] << std::setw(5);
		}
		cout << "\n";
	}
	cout << "\n";

	// CMO	=>	mat_prod_T	:= alpha * mat_prod.T  +  beta * mat_prod.T
	//						:= mat_prod.T
	status = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, r, p, &alpha, mat_prod, p, &beta, mat_prod, p, mat_prod_T, r);
	hipDeviceSynchronize();
	for (size_t i = 0; i < p; i++)
	{
		for (size_t j = 0; j < r; j++)
		{
			cout << mat_prod_T[i * r + j] << std::setw(5);
		}
		cout << "\n";
	}


	// Clean up the created handle
	hipblasDestroy(handle);

	// Release allocated memory
	hipFree(mat1);
	hipFree(mat2);
	hipFree(mat_prod);
	hipFree(mat_prod_T);

	cout << "\n\n";
}


/**
 * Implemented in RMO
 * C = AX  ;   A(mxn) ; X(nx1)
 * https://www.adityaagrawal.net/blog/deep_learning/row_column_major
 * https://stackoverflow.com/questions/14595750/transpose-matrix-multiplication-in-cublas-howto
 */
void axpb_vector_matrix_() {

	size_t m = 5, n = 3;
	float* A; // m x n
	float* X; // n x 1
	float b = 1; // scalar
	float* C; // m x 1
	float alpha = 1, beta = 0;

	hipMallocManaged(&A, m * n * sizeof(float));
	hipMallocManaged(&X, n * 1 * sizeof(float));
	hipMallocManaged(&C, m * 1 * sizeof(float));

	// Init A
	int val = 0;
	for (size_t i = 0; i < m; i++)
	{
		for (size_t j = 0; j < n; j++)
		{
			A[i * n + j] = val;
			val = (val + 1) % 8;
		}
	}
	cout << "\n";
	for (size_t i = 0; i < m; i++)
	{
		for (size_t j = 0; j < n; j++)
		{
			cout << A[i * n + j] << std::setw(5);
		}
		cout << "\n";
	}

	// Init X
	val = 1;
	for (size_t i = 0; i < n; i++)
	{
		X[i] = val;
		val = (val + 1);
	}
	cout << "\n";
	for (size_t i = 0; i < n; i++)
	{
		cout << X[i] << "\n";
	}

	// Init C
	for (size_t i = 0; i < m; i++)
	{
		C[i] = 0;
	}
	cout << "\n";
	for (size_t i = 0; i < m; i++)
	{
		cout << C[i] << "\n";
	}

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemv(handle, HIPBLAS_OP_T, n, m, &alpha, A, n, X, 1, &beta, C, 1 );
	hipDeviceSynchronize();

	cout << "\n";
	for (size_t i = 0; i < m; i++)
	{
		cout << C[i] << "\n";
	}

	hipblasDestroy(handle);
	hipFree(A);
	hipFree(X);
	hipFree(C);
}


int main_1() {
	//mat_mul();
	//mat_mul_2();
	//axpy();
	//axpb_vector_matrix();
	axpb_vector_matrix_();
	//map();
	//transpose_mat();
	return 0;
}