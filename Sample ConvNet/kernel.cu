
#include "hip/hip_runtime.h"


 // need CUDA_VERSION
#include <cudnn.h>

#include <stdio.h>


int main()
{
    int version = (int)cudnnGetVersion();
    printf("cudnnGetVersion() : %d , CUDNN_VERSION from cudnn.h : %d \n", version, CUDNN_VERSION);

}
